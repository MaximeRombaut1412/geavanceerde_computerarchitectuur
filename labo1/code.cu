
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

int main()
{
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

  // Allocate the host input vector A
  float *h_A = (float *)malloc(size);

  // Allocate the host input vector B
  float *h_B = (float *)malloc(size);

  // Allocate the host output vector C
  float *h_C = (float *)malloc(size);

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) 
  {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate the device input vector A
  float *d_A = NULL;

  // Allocate the device input vector B
  float *d_B = NULL;

  // Allocate the device output vector C
  float *d_C = NULL;

  printf("Copy input data from the host memory to the CUDA device\n");

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

  printf("Copy output data from the CUDA device to the host memory\n");

  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

    return 0;
}