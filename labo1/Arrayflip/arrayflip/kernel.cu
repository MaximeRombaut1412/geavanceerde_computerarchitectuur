#include "hip/hip_runtime.h"


#include <stdio.h>

int main()
{
    //initialize arrays
    const int SIZE = 128000;
    int A[SIZE];
    int B[sizeof(A) / sizeof(int)] = { 0 };

    // fill array A
    for (int i = 0; i < SIZE; i++) {

        A[i] = rand();
    }

    //flip array A into array B
    for (int i = 0; i < SIZE; i++)
    {
        B[i] = A[SIZE - 1 - i];
    }

    // Verify that the result vector is correct
    for (int i = 0; i < SIZE; ++i)
    {
        if (B[i] != A[SIZE - i - 1])
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    return 0;
}