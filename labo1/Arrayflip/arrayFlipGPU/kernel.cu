#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void arrayFlip(int* input, int* output, int numElements)
{
    int i = threadIdx.x;
    output[numElements - i - 1] = input[i];
}

int main()
{
    //initialize arrays
    const int SIZE = 128000;
    int A[SIZE];
    int B[sizeof(A) / sizeof(int)] = { 0 };

    // fill array A
    for (int i = 0; i < SIZE; i++) {

        A[i] = rand();
    }

    //pointers to GPU memory
    int* gpuA = 0;
    int* gpuB = 0;

    //allocate memory for arrays
    hipMalloc(&gpuA, sizeof(A));
    hipMalloc(&gpuB, sizeof(B));

    //copy arrays to GPU
    hipMemcpy(gpuA, A, sizeof(A), hipMemcpyHostToDevice);

    //call GPU function
    arrayFlip <<< 1, sizeof(a)/sizeof(int) >>> (cudaA, cudaB, SIZE);

    //copy result into B
    hipMemcpy(B, gpuB, sizeof(B), hipMemcpyDeviceToHost);

    return 0;
}